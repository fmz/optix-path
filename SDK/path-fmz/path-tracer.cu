#include "hip/hip_runtime.h"
#include <optix.h>
#include <cuda/helpers.h>

//#include "sutil/vec_math.h"

#include "path-tracer.h"
#include "random.h"

extern "C" {
__constant__ Params params;


struct OrthonormalBasis {
    __forceinline__ __device__ OrthonormalBasis(const float3& normal) {
        m_normal = normal;

        float x = normal.x, y = normal.y, z = normal.z;

        // Lifted from pbrt
        float sign = z >= 0.f ? 1.f : -1.f;
        float a = -1.f / (sign + z);
        float b = x * y * a;

        m_binormal = {b, sign + y*y, -y};
        m_binormal = normalize(m_binormal);
        m_tangent  = cross(m_binormal, m_normal);
    }

    __forceinline__ __device__ void inverse_transform(float3& p) const {
        p = p.x*m_tangent + p.y*m_binormal + p.z*m_normal;
    }

    float3 m_tangent;
    float3 m_binormal;
    float3 m_normal;
};

// Utility functions

static __forceinline__ __device__ RadiancePRD loadClosesthitRadiancePRD()
{
    RadiancePRD prd = {};

    prd.attenuation.x = __uint_as_float(optixGetPayload_0());
    prd.attenuation.y = __uint_as_float(optixGetPayload_1());
    prd.attenuation.z = __uint_as_float(optixGetPayload_2());
    prd.seed  = optixGetPayload_3();
    prd.depth = optixGetPayload_4();
    return prd;
}

static __forceinline__ __device__ RadiancePRD loadMissRadiancePRD() {
    RadiancePRD prd = {};
    return prd;
}

static __forceinline__ __device__ void storeClosesthitRadiancePRD(RadiancePRD prd) {
    optixSetPayload_0(__float_as_uint(prd.attenuation.x));
    optixSetPayload_1(__float_as_uint(prd.attenuation.y));
    optixSetPayload_2(__float_as_uint(prd.attenuation.z));

    optixSetPayload_3(prd.seed);
    optixSetPayload_4(prd.depth);

    optixSetPayload_5(__float_as_uint(prd.emitted.x));
    optixSetPayload_6(__float_as_uint(prd.emitted.y));
    optixSetPayload_7(__float_as_uint(prd.emitted.z));

    optixSetPayload_8(__float_as_uint(prd.radiance.x));
    optixSetPayload_9(__float_as_uint(prd.radiance.y));
    optixSetPayload_10(__float_as_uint(prd.radiance.z));

    optixSetPayload_11(__float_as_uint(prd.origin.x));
    optixSetPayload_12(__float_as_uint(prd.origin.y));
    optixSetPayload_13(__float_as_uint(prd.origin.z));

    optixSetPayload_14(__float_as_uint(prd.direction.x));
    optixSetPayload_15(__float_as_uint(prd.direction.y));
    optixSetPayload_16(__float_as_uint(prd.direction.z));

    optixSetPayload_17(prd.done);
}

static __forceinline__ __device__ void storeMissRadiancePRD(RadiancePRD prd) {
    optixSetPayload_5(__float_as_uint(prd.emitted.x));
    optixSetPayload_6(__float_as_uint(prd.emitted.y));
    optixSetPayload_7(__float_as_uint(prd.emitted.z));

    optixSetPayload_8(__float_as_uint(prd.radiance.x));
    optixSetPayload_9(__float_as_uint(prd.radiance.y));
    optixSetPayload_10(__float_as_uint(prd.radiance.z));

    optixSetPayload_17(prd.done);
}

static __forceinline__ __device__ void cosine_sample_hemisphere(const float u1, const float u2, float3& p)
{
    // Uniformly sample disk.
    const float r   = sqrtf( u1 );
    const float phi = 2.0f*M_PIf * u2;
    p.x = r * cosf( phi );
    p.y = r * sinf( phi );

    // Project up to hemisphere.
    p.z = sqrtf( fmaxf( 0.0f, 1.0f - p.x*p.x - p.y*p.y ) );
}


// static __forceinline__ __device__ void setPayload(float3 p) {
//     optixSetPayload_0(__float_as_uint(p.x));
//     optixSetPayload_1(__float_as_uint(p.y));
//     optixSetPayload_2(__float_as_uint(p.z));
//}

// static __forceinline__ __device__ void computeRay(uint3 idx, uint3 dim, float3& origin, float3& direction) {
//     const float3 U = params.cam_u;
//     const float3 V = params.cam_v;
//     const float3 W = params.cam_w;

//     const float2 d = 2.f * make_float2(
//         static_cast<float>(idx.x) / static_cast<float>(dim.x),
//         static_cast<float>(idx.y) / static_cast<float>(dim.y)
//         ) - 1.f;

//     origin    = params.cam_eye;
//     direction = normalize(d.x * U + d.y * V + W);
// }

static __forceinline__ __device__ void traceRadiance(
    OptixTraversableHandle handle,
    float3                 ray_o,
    float3                 ray_d,
    float                  tmin,
    float                  tmax,
    RadiancePRD&           prd
) {
    unsigned int u0, u1, u2, u3, u4, u5, u6, u7, u8, u9, u10, u11, u12, u13, u14, u15, u16, u17;

    u0 = __float_as_uint(prd.attenuation.x);
    u1 = __float_as_uint(prd.attenuation.y);
    u2 = __float_as_uint(prd.attenuation.z);
    u3 = prd.seed;
    u4 = prd.depth;

    optixTraverse(
        PAYLOAD_TYPE_RADIANCE,
        handle,
        ray_o,
        ray_d,
        tmin,
        tmax,
        0.0f,                     // rayTime
        OptixVisibilityMask(1),
        OPTIX_RAY_FLAG_NONE,
        0,                        // SBT offset
        RAY_TYPE_COUNT,           // SBT stride
        0,                        // missSBTIndex
        u0, u1, u2, u3, u4, u5, u6, u7, u8, u9, u10, u11, u12, u13, u14, u15, u16, u17
    );

    optixInvoke(PAYLOAD_TYPE_RADIANCE,
        u0, u1, u2, u3, u4, u5, u6, u7, u8, u9, u10, u11, u12, u13, u14, u15, u16, u17
    );

    prd.attenuation = make_float3(__uint_as_float(u0), __uint_as_float(u1), __uint_as_float(u2));
    prd.seed  = u3;
    prd.depth = u4;

    prd.emitted   = make_float3(__uint_as_float(u5),  __uint_as_float(u6),  __uint_as_float(u7));
    prd.radiance  = make_float3(__uint_as_float(u8),  __uint_as_float(u9),  __uint_as_float(u10));
    prd.origin    = make_float3(__uint_as_float(u11), __uint_as_float(u12), __uint_as_float(u13));
    prd.direction = make_float3(__uint_as_float(u14), __uint_as_float(u15), __uint_as_float(u16));
    prd.done      = u17;
}

// Returns true if ray is occluded, else false
static __forceinline__ __device__ bool traceOcclusion(
    OptixTraversableHandle handle,
    float3                 ray_origin,
    float3                 ray_direction,
    float                  tmin,
    float                  tmax
) {
    // We are only casting probe rays so no shader invocation is needed
    optixTraverse(
        handle,
        ray_origin,
        ray_direction,
        tmin,
        tmax, 0.0f,                // rayTime
        OptixVisibilityMask(1),
        OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT | OPTIX_RAY_FLAG_DISABLE_ANYHIT,
        0,                         // SBT offset
        RAY_TYPE_COUNT,            // SBT stride
        0                          // missSBTIndex
    );
    return optixHitObjectIsHit();
}

__global__ void __raygen__path_tracer() {
    const int32_t w   = params.width;
    const int32_t h   = params.height;
    const float3  eye = params.cam_eye;
    const float3  U   = params.cam_u;
    const float3  V   = params.cam_v;
    const float3  W   = params.cam_w;
    const float cont_prob = params.continuation_prob;

    const uint3   idx          = optixGetLaunchIndex();
    const int32_t subframe_idx = params.subframe_index;
    // const uint3 dim = optixGetLaunchDimensions();

    // TODO: use a better RNG
    uint32_t seed = tea<4>(idx.y*w + idx.x, subframe_idx);

    float3 result = {0.f, 0.f, 0.f};
    int32_t spl = params.samples_per_launch;
    for (int32_t i = 0; i < spl; i++) {
        // Map thread id to screen coords, and shoot a ray out (in world coords)

        // Vary the target pixel by 0.5 in each direction when mapping to screen space.
        const float2 subpixel_offset = {rnd(seed), rnd(seed)};
        const float2 dir = 2.f * make_float2(
            (float(idx.x) + subpixel_offset.x) / float(w),
            (float(idx.y) + subpixel_offset.y) / float(h)
        ) - 1.f;

        float3 ray_o = eye;
        float3 ray_d = normalize(dir.x * U + dir.y * V + W);

        RadiancePRD prd;
        prd.attenuation = {1.f, 1.f, 1.f};
        prd.seed        = seed;
        prd.depth       = 0;

        while (true) {
            // Trace ray!
            traceRadiance(
                params.handle,
                ray_o,
                ray_d,
                0.0001f,  // tmin
                1e16f,  // tmax
                prd
            );
            result += prd.emitted;
            result += prd.radiance * prd.attenuation;

            const float p = dot(prd.attenuation, {0.3f, 0.59f, 0.11f});
            const bool done = prd.done || rnd(prd.seed) > p;

            // Russian Roulette
            // const float p = rnd(prd.seed);
            // const bool done = prd.done || (p > cont_prob);
            if (done) {
                break;
            }
            prd.attenuation /= p;

            ray_o = prd.origin;
            ray_d = prd.direction;

            prd.depth++;
        }
    }

    const uint32_t image_idx    = idx.y * w + idx.x;
    float3         accum_color  = result / static_cast<float>(params.samples_per_launch); // TODO: use a filter instead

    if(subframe_idx > 0) {
        const float                 a = 1.0f / static_cast<float>( subframe_idx+1 );
        const float3 accum_color_prev = make_float3(params.accum_buffer[image_idx]);
        accum_color = lerp(accum_color_prev, accum_color, a);
    }
    params.accum_buffer[image_idx] = make_float4(accum_color, 1.0f);
    params.frame_buffer[image_idx] = make_color (accum_color);
}

__global__ void __miss__radiance()
{
    optixSetPayloadTypes(PAYLOAD_TYPE_RADIANCE);

    MissData* rt_data = reinterpret_cast<MissData*>(optixGetSbtDataPointer());
    RadiancePRD prd   = loadMissRadiancePRD();

    prd.radiance  = make_float3(rt_data->bg_clr);
    prd.emitted   = make_float3(0.f);
    prd.done      = true;

    storeMissRadiancePRD(prd);
}


static __forceinline__ __device__ float3 getNormal(const HitGroupData* rt_data, int32_t vert_idx) {
    const float3 n1  = normalize(rt_data->normals[vert_idx+0]);
    // if the
    const float3 n2  = normalize(rt_data->normals[vert_idx+1]);
    const float3 n3  = normalize(rt_data->normals[vert_idx+2]);

    // Get barycentric coords
    const float2 bc = optixGetTriangleBarycentrics();
    float v = bc.x;
    float w = bc.y;
    float u = 1.f - v - w;

    float3 interpolated_normal = normalize(u * n1 + v * n2 + w * n3);
    return interpolated_normal;
}

__global__ void __closesthit__radiance() {
    optixSetPayloadTypes(PAYLOAD_TYPE_RADIANCE);

    HitGroupData* rt_data = (HitGroupData*)optixGetSbtDataPointer();

    const int    prim_idx        = optixGetPrimitiveIndex();
    const float3 ray_dir         = optixGetWorldRayDirection();
    const int    vert_idx_offset = prim_idx*3;

    const float3 n    = getNormal(rt_data, vert_idx_offset);

    const float3 N    = n;//aceforward(n, -ray_dir, n);
    const float3 P    = optixGetWorldRayOrigin() + optixGetRayTmax()*ray_dir;

    RadiancePRD prd = loadClosesthitRadiancePRD();

    if(prd.depth == 0) {
        prd.emitted = rt_data->emission_color;
    } else {
        prd.emitted = make_float3( 0.0f );
    }

    unsigned int seed = prd.seed;
    {
        const float z1 = rnd(seed);
        const float z2 = rnd(seed);

        float3 w_in;
        cosine_sample_hemisphere(z1, z2, w_in);
        OrthonormalBasis onb(N);
        onb.inverse_transform(w_in);
        prd.direction = w_in;
        prd.origin    = P;

        prd.attenuation *= rt_data->diffuse_color;
    }

    const float z1 = rnd(seed);
    const float z2 = rnd(seed);
    prd.seed = seed;

    ParallelogramLight light = params.light;
    const float3 light_pos   = light.corner + light.v1 * z1 + light.v2 * z2;

    // Calculate properties of light sample (for area based pdf)
    const float  Ldist = length(light_pos - P );
    const float3 L     = normalize(light_pos - P );
    const float  nDl   = dot( N, L );
    const float  LnDl  = -dot( light.normal, L );

    float weight = 0.0f;
    if( nDl > 0.0f && LnDl > 0.0f ) {
        const bool occluded =
            traceOcclusion(
                params.handle,
                P,
                L,
                0.0001f,           // tmin
                Ldist - 0.0001f);  // tmax

        if( !occluded ) {
            const float A = length(cross(light.v1, light.v2));
            weight = nDl * LnDl * A / (M_PIf * Ldist * Ldist);
        }
    }

    prd.radiance = light.emission * weight;
    prd.done     = false;

    storeClosesthitRadiancePRD( prd );
}
// __global__ void __miss__test_shader() {
//     MissData* miss_data = reinterpret_cast<MissData*>(optixGetSbtDataPointer());
//     setPayload(miss_data->bg_clr);
// }

// __global__ void __closesthit__test_shader() {
//     // Neat stuff! we can do barycentrics here.
//     const float2 barycentrics = optixGetTriangleBarycentrics();
//     setPayload(make_float3(barycentrics, 1.f));
// }

}  // extern "C"

